/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
 #include "BatchData.h"
 #include "zlib.h"
 #include "libdeflate.h"
 #include "nvcomp/deflate.h"

 // Benchmark performance from the binary data file fname
 static void run_example(const std::vector<std::vector<char>>& data, int algo)
 {
   size_t total_bytes = 0;
   for (const std::vector<char>& part : data) {
     total_bytes += part.size();
   }
 
   std::cout << "----------" << std::endl;
   std::cout << "files: " << data.size() << std::endl;
   std::cout << "uncompressed (B): " << total_bytes << std::endl;
 
   const size_t chunk_size = 1 << 16;
 
   // build up input batch on CPU
   BatchDataCPU input_data_cpu(data, chunk_size);
   std::cout << "chunks: " << input_data_cpu.size() << std::endl;
 
   // compression
 
   // Allocate and prepare output/compressed batch
   BatchDataCPU compress_data_cpu(
       chunk_size, input_data_cpu.size());
 
   // loop over chunks on the CPU, compressing each one
   for (size_t i = 0; i < input_data_cpu.size(); ++i) {
     int actual_len = 0;
     if(algo==0){ //libdeflate
       struct libdeflate_compressor *compressor;
       compressor = libdeflate_alloc_compressor(6%13);
       int len = libdeflate_deflate_compress(compressor, input_data_cpu.ptrs()[i],  
                             input_data_cpu.sizes()[i], compress_data_cpu.ptrs()[i], compress_data_cpu.sizes()[i]);
       if (len == 0) {
         throw std::runtime_error(
             "libdeflate_deflate_compress failed to compress chunk " + std::to_string(i) + ".");
       }
       actual_len = len;
     }else if(algo==1){ //zlib::compress2
      uLongf len = input_data_cpu.sizes()[i];
      int ret = compress2((uint8_t *)compress_data_cpu.ptrs()[i], &len, (const Bytef *) input_data_cpu.ptrs()[i], input_data_cpu.sizes()[i], 9);
      if (ret != Z_OK) {
          throw std::runtime_error("ZLIB compress() failed " + std::to_string(ret));
      }
      if (len >= 6) {
        memmove((uint8_t*)compress_data_cpu.ptrs()[i], (uint8_t*)compress_data_cpu.ptrs()[i] + 2, len - 6);
        len -= 6;
      }
      actual_len = len;
     }else if(algo==2){ //zlib::deflate
      z_stream zs;
      zs.zalloc = NULL; zs.zfree = NULL;
      zs.msg = NULL;
      zs.next_in  = (Bytef *)input_data_cpu.ptrs()[i];
      zs.avail_in = input_data_cpu.sizes()[i];
      zs.next_out = (Bytef *)compress_data_cpu.ptrs()[i];
      zs.avail_out = input_data_cpu.sizes()[i];
      int strategy=Z_DEFAULT_STRATEGY; //Z_HUFFMAN_ONLY //Z_FIXED, Z_DEFAULT_STRATEGY 
      int ret = deflateInit2(&zs, 9, Z_DEFLATED, -15, 8, strategy                       ); // -15 to disable zlib header/footer
      if (ret!=Z_OK) {
          throw std::runtime_error("Call to deflateInit2 failed: " + std::to_string(ret));
      }
      if ((ret = deflate(&zs, Z_FINISH)) != Z_STREAM_END) {
          throw std::runtime_error("Deflate operation failed: " + std::to_string(ret));
      }
      if ((ret = deflateEnd(&zs)) != Z_OK) {
          throw std::runtime_error("Call to deflateEnd failed: " + std::to_string(ret));
      }
      actual_len = zs.total_out;
     }
    // set the actual compressed size
    compress_data_cpu.sizes()[i] = actual_len;
   }
 
   // compute compression ratio
   size_t* compressed_sizes_host = compress_data_cpu.sizes();
   size_t comp_bytes = 0;
   for (size_t i = 0; i < compress_data_cpu.size(); ++i)
     comp_bytes += compressed_sizes_host[i];
 
   std::cout << "comp_size: " << comp_bytes
             << ", compressed ratio: " << std::fixed << std::setprecision(2)
             << (double)total_bytes / comp_bytes << std::endl;
 
   // Copy compressed data to GPU
   BatchData compress_data(compress_data_cpu, true);
 
   // Allocate and build up decompression batch on GPU
   BatchData decomp_data(input_data_cpu, false);
 
   // Create CUDA stream
   hipStream_t stream;
   hipStreamCreate(&stream);
 
   // CUDA events to measure decompression time
   hipEvent_t start, end;
   hipEventCreate(&start);
   hipEventCreate(&end);

   // deflate GPU decompression
   size_t decomp_temp_bytes;
   nvcompStatus_t status = nvcompBatchedDeflateDecompressGetTempSize(
       compress_data.size(), chunk_size, &decomp_temp_bytes);
   if (status != nvcompSuccess) {
     throw std::runtime_error("nvcompBatchedDeflateDecompressGetTempSize() failed.");
   }
 
   void* d_decomp_temp;
   CUDA_CHECK(hipMalloc(&d_decomp_temp, decomp_temp_bytes));
 
   size_t* d_decomp_sizes;
   CUDA_CHECK(hipMalloc(&d_decomp_sizes, decomp_data.size() * sizeof(size_t)));
 
   nvcompStatus_t* d_status_ptrs;
   CUDA_CHECK(hipMalloc(&d_status_ptrs, decomp_data.size() * sizeof(nvcompStatus_t)));
 
   CUDA_CHECK(hipStreamSynchronize(stream));
 
   // Run decompression
   status = nvcompBatchedDeflateDecompressAsync(
       compress_data.ptrs(),
       compress_data.sizes(),
       decomp_data.sizes(),
       d_decomp_sizes,
       compress_data.size(),
       d_decomp_temp,
       decomp_temp_bytes,
       decomp_data.ptrs(),
       d_status_ptrs,
       stream);
   if( status != nvcompSuccess){
     throw std::runtime_error("ERROR: nvcompBatchedDeflateDecompressAsync() not successful");
   }
 
   // Validate decompressed data against input
   if (!(input_data_cpu == decomp_data))
     throw std::runtime_error("Failed to validate decompressed data");
   else
     std::cout << "decompression validated :)" << std::endl;
 
   // Re-run decompression to get throughput
   hipEventRecord(start, stream);
   status = nvcompBatchedDeflateDecompressAsync(
     compress_data.ptrs(),
     compress_data.sizes(),
     decomp_data.sizes(),
     d_decomp_sizes,
     compress_data.size(),
     d_decomp_temp,
     decomp_temp_bytes,
     decomp_data.ptrs(),
     d_status_ptrs,
     stream);
   hipEventRecord(end, stream);
   if( status != nvcompSuccess){
     throw std::runtime_error("ERROR: nvcompBatchedDeflateDecompressAsync() not successful");
   }
 
   CUDA_CHECK(hipStreamSynchronize(stream));
 
   float ms;
   hipEventElapsedTime(&ms, start, end);
 
   double decompression_throughput = ((double)total_bytes / ms) * 1e-6;
   std::cout << "decompression throughput (GB/s): " << decompression_throughput
             << std::endl;
 
   hipFree(d_decomp_temp);
 
   hipEventDestroy(start);
   hipEventDestroy(end);
   hipStreamDestroy(stream);
 }
 
 std::vector<char> readFile(const std::string& filename)
 {
   std::vector<char> buffer(4096);
   std::vector<char> host_data;
 
   std::ifstream fin(filename, std::ifstream::binary);
   fin.exceptions(std::ifstream::failbit | std::ifstream::badbit);
 
   size_t num;
   do {
     num = fin.readsome(buffer.data(), buffer.size());
     host_data.insert(host_data.end(), buffer.begin(), buffer.begin() + num);
   } while (num > 0);
 
   return host_data;
 }
 
 std::vector<std::vector<char>>
 multi_file(const std::vector<std::string>& filenames)
 {
   std::vector<std::vector<char>> split_data;
 
   for (auto const& filename : filenames) {
     split_data.emplace_back(readFile(filename));
   }
 
   return split_data;
 }
 
 int main(int argc, char* argv[])
 {
   std::vector<std::string> file_names;
 
   if (argc < 5) {
     std::cerr << "Must choose the algorithm (-a <0>) and specify at least one file (-f <inputfile>)." << std::endl;
     return 1;
   }
   int algo = 0;
   int i = 1; bool choose_algo = false; bool input_file = false;
   do{
    if(strcmp(argv[i], "-a") !=0 && strcmp(argv[i], "-f") != 0){
      std::cerr << "The config only could be -a (choose algorithm: 0 libdeflate, 1 zlib_compress2, 2 zlib_deflate) or -f (add input files)." << std::endl;
      return 1;
    }else if(strcmp(argv[i], "-a") ==0){
      choose_algo = true;
      i++;
      if( (i < argc) && (atoi(argv[i]) == 0 ||  atoi(argv[i]) == 1 || atoi(argv[i]) == 2)){
        algo = atoi(argv[2]);
        i++;
      }else{
        std::cerr<<"`-a` could only be 0, 1, 2. (0 libdeflate, 1 zlib_compress2, 2 zlib_deflate)"<<std::endl;
        return 1;
      }
    }else if (strcmp(argv[i], "-f") == 0){
      i++;
      if(i >= argc){
        std::cerr<<"Specify at least one input file." <<std::endl;
        return 1;
      }
      do{
        input_file = true;
        file_names.push_back(argv[i]);
        i++;
      }while(i < argc && strcmp(argv[i], "-a") !=0);
    }
   }while(i < argc);

   if(!choose_algo){
    std::cerr<<"Have to choose an algorithm use `-a`. `-a` could be 0, 1, 2. (0 libdeflate, 1 zlib_compress2, 2 zlib_deflate)"<<std::endl;
    return 1;
   }

   if(!input_file){
    std::cerr<<"Specify at least one input file by using `-f`"<<std::endl;
    return 1;
   }

   auto data = multi_file(file_names);
   run_example(data, algo);
 
   return 0;
 }
 